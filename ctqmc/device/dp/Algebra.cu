#include "hip/hip_runtime.h"
#include <cstring>
#include <type_traits>
#include <sstream>

#include "Algebra.h"
#include "Variant.h"

#include "../include/Errchk.h"
#include "../include/Allocator.h"

#include "../../../include/BlasLapack.h"
#include "../../../include/mpi/Basic.h"


#ifdef HAVE_CUBLAS

#include <hipblas.h>

__device__ double deviceZero = .0;
__device__ double deviceOne  = 1.;
__device__ hipblasHandle_t deviceHandle;

__global__ void kerCublasHandle(int const flag) {
    flag ? hipblasCreate(&deviceHandle) : hipblasDestroy(deviceHandle);
};

#else

#include <cutlass/gemm/gemm.h>
#include <cutlass/gemm/dgemm_traits.h>
#include <cutlass/gemm/zgemm_traits.h>

#endif


using namespace imp;
using namespace device;


constexpr int WarpSize = 32;  Allocator* alloc = nullptr;


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int imp::pci_id_size() {
    return 16;
}

// @#$!#$@!# cudaGetDevicePciBusId is not working properly on SummitDev ......
void get_pci_id(char* pci_id, int deviceId) {
    std::stringstream stream;
    hipDeviceProp_t deviceProperties; cudaErrchk(hipGetDeviceProperties(&deviceProperties, deviceId));
    stream << std::hex << deviceProperties.pciDomainID << ":" << deviceProperties.pciBusID << ":" << deviceProperties.pciDeviceID;
    std::string str = stream.str(); std::copy(str.begin(), str.end(), pci_id);
}


int imp::get_pci_ids(std::vector<char>& pciIds) {
    int deviceCount; cudaErrchk(hipGetDeviceCount(&deviceCount));
    
    pciIds.resize(deviceCount*pci_id_size(), '\0');
    for(int id = 0; id < deviceCount; ++id) get_pci_id(&pciIds[id*pci_id_size()], id);
    
    return deviceCount;
}

void imp::init_device(std::vector<char> const& pciId, std::size_t processesPerDevice) {
    int deviceId; cudaErrchk(hipDeviceGetByPCIBusId(&deviceId, pciId.data()));
    cudaErrchk(hipSetDevice(deviceId));
    
    hipDeviceProp_t deviceProperties; cudaErrchk(hipGetDeviceProperties(&deviceProperties, deviceId));
    if(deviceProperties.computeMode != hipComputeModeExclusive && deviceProperties.computeMode != hipComputeModeExclusiveProcess)
        throw std::runtime_error("Please set GPU compute mode to \"hipComputeModeExclusive\" or \"hipComputeModeExclusiveProcess\"");
    if(deviceProperties.warpSize != WarpSize)
        throw std::runtime_error("Please set WarpSize in AlgebraDevice.cu to " + std::to_string(deviceProperties.warpSize));
    
    cudaErrchk(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
    
#ifdef HAVE_CUBLAS
    
    kerCublasHandle<<<1, 1>>>(1);
    
#endif
    
    alloc = new Allocator((0.8/processesPerDevice)*deviceProperties.totalGlobalMem);
    
    cudaErrchk(hipDeviceSynchronize());
}

void imp::release_device() {
    if(!alloc->sanity_check()) throw std::runtime_error("Memory leak !");
    
    delete alloc;
    alloc = nullptr;
    
#ifdef HAVE_CUBLAS
    
    kerCublasHandle<<<1, 1>>>(0);
    
#endif
    
    cudaErrchk(hipDeviceSynchronize());
};


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//-------------------------------------------------------------------------------------------------------------------------------------------------

imp::Energies<Device>::Energies(jsx::value const& jParams, std::vector<double> const& energies) :
dim0_(energies.size()),
dim_(jParams.is("trunc dim") ? std::min<int>(dim0_, jParams("trunc dim").int64()) : dim0_),
ln_dim_(std::log(dim_)),
data_(alloc->get<double>(dim_)),
min_(std::numeric_limits<double>::max()) {
    for(int i = 0; i < dim_; ++i)
        min_ = std::min(min_, energies[i]);
    
    cudaErrchk(hipMemcpy(data_.ptr(), energies.data(), dim_*sizeof(double), hipMemcpyHostToDevice));
}
imp::Energies<Device>::~Energies() {
    alloc->free(data_);
}

//-------------------------------------------------------------------------------------------------------------------------------------------------

imp::Vector<Device>::Vector(double time, Energies<Device> const& energies) :
time_(time),
exponent_(time*energies.min()),
energies_(energies) {
}
imp::Vector<Device>::~Vector() {
}

//-------------------------------------------------------------------------------------------------------------------------------------------------
template <typename Value>
imp::Matrix<Device, Value>::Matrix(int size) : data_(alloc->get<cuda_value_trait_t<Value>>(size)) {
}

template <typename Value>
imp::Matrix<Device, Value>::Matrix(Matrix<Device, Value>::Identity const& identity) : I_(identity.dim), J_(identity.dim), data_(alloc->get<cuda_value_trait_t<Value>>(I_*J_)), exponent_(.0) {
    Value* temp = new Value[I_*J_]; std::memset(temp, 0, I_*J_*sizeof(Value));
    for(int i = 0; i < identity.dim; ++i) temp[i*(identity.dim + 1)] = 1.; //kack memset isch das allgemein für double's ?
    
    cudaErrchk(hipMemcpy(data_.ptr(), temp, I_*J_*sizeof(Value), hipMemcpyHostToDevice));
    
    delete[] temp;
}

template <typename Value>
imp::Matrix<Device, Value>::Matrix(Matrix<Device, Value>::Zero const& zero) : I_(zero.dim), J_(zero.dim), data_(alloc->get<cuda_value_trait_t<Value>>(I_*J_)), exponent_(.0) {
    Value* temp = new Value[I_*J_]; std::memset(temp, 0, I_*J_*sizeof(Value));
    
    cudaErrchk(hipMemcpy(data_.ptr(), temp, I_*J_*sizeof(Value), hipMemcpyHostToDevice));
    
    delete[] temp;
}

template <typename Value>
imp::Matrix<Device, Value>::Matrix(int I, int J, io::Matrix<Value> const& mat) : I_(I), J_(J), data_(alloc->get<cuda_value_trait_t<Value>>(I_*J_)), exponent_(.0) {
    Value* temp = new Value[I_*J_];
    
    for(int i = 0; i < I; ++i)
        for(int j = 0; j < J; ++j)
            temp[j + J*i] = mat(i, j);
    
    cudaErrchk(hipMemcpy(data_.ptr(), temp, I_*J_*sizeof(Value), hipMemcpyHostToDevice));
    
    delete[] temp;
}

template <typename Value>
imp::Matrix<Device, Value>::~Matrix() {
    alloc->free(data_);
}

//-------------------------------------------------------------------------------------------------------------------------------------------------

template <>
void imp::add<double>(double* dest, double fact, Matrix<Device, double> const& source) {
    int const N = source.I()*source.J(); int const one = 1;
    double* temp = new double[N];                              //Ja scheisse das isch beschisse, passiert aber nit oft.
    
    cudaErrchk(hipMemcpy(temp, source.data().ptr(), N*sizeof(double), hipMemcpyDeviceToHost));
    daxpy_(&N, &fact, temp, &one, dest, &one);
    
    delete[] temp;
}

template <>
void imp::add<ut::complex>(ut::complex* dest, ut::complex fact, Matrix<Device, ut::complex> const& source) {
    int const N = source.I()*source.J(); int const one = 1;
    ut::complex* temp = new ut::complex[N];                              //Ja scheisse das isch beschisse, passiert aber nit oft.
    
    cudaErrchk(hipMemcpy(temp, source.data().ptr(), N*sizeof(ut::complex), hipMemcpyDeviceToHost));
    zaxpy_(&N, &fact, temp, &one, dest, &one);
    
    delete[] temp;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//-------------------------------------------------------------------------------------------------------------------------------------------------


template <typename Value>
struct CopyEvolveL {
    double time;
    double shift;
    double const* energies;
    cuda_value_trait_t<Value> const* source;
    cuda_value_trait_t<Value>* dest;
    int I;
    int J;
};


template <typename Value>
__global__ void kerCopyEvolveL(CopyEvolveL<Value> args) {
    int const i = blockIdx.x; int const j = threadIdx.x;

    args.dest[j + blockDim.x*i] = exp(args.time*args.energies[i] - args.shift)*args.source[j + blockDim.x*i];
};


template <typename Value>
void imp::copyEvolveL(Matrix<Device, Value>& dest, Vector<Device> const& prop, Matrix<Device, Value> const& source, itf::Batcher<Value>& batcher) {
    dest.I() = source.I(); dest.J() = source.J(); dest.exponent() = source.exponent() + prop.exponent(); // eigentli source.exponent() = 0, isch aber sicherer so
    
    auto& args = imp::get<Device>(batcher).template get_kernel<CopyEvolveL<Value>>();
    
    args.time     = prop.time();
    args.shift    = prop.exponent();
    args.energies = prop.energies().data().ptr();
    args.source   = source.data().ptr();
    args.dest     = dest.data().ptr();
    args.I        = source.I();
    args.J        = source.J();
}

//-------------------------------------------------------------------------------------------------------------------------------------------------

template <typename Value>
struct Mult {
    cuda_value_trait_t<Value> const* A;
    cuda_value_trait_t<Value> const* B;
    cuda_value_trait_t<Value>* C;
    int M;
    int N;
    int K;
};

#ifndef HAVE_CUBLAS

template <typename KernelClass>
__global__ void cutlass_kernel(typename KernelClass::Params const& params)
{
    extern __shared__ int GemmSharedStorageBase[];
    
    typename KernelClass::SharedStorage *shared_storage =
    reinterpret_cast<typename KernelClass::SharedStorage *>(GemmSharedStorageBase);
    
    KernelClass gemm(params, *shared_storage);
    
    gemm.multiply_add();
}

template<typename BlockShape, typename ThreadShape>
__device__ void cutlass_gemm(Mult<double> const& args, Byte*& memory)
{
    typedef cutlass::gemm::DgemmTraits<
    cutlass::MatrixLayout::kColumnMajor,   // layout of A matrix
    cutlass::MatrixLayout::kColumnMajor,
    BlockShape,
    cutlass::gemm::LinearScaling<double>,
    ThreadShape
    >
    Traits;
    
    typedef typename Traits::Params Params;
    typedef typename Traits::KernelClass KernelClass;

    memory = reinterpret_cast<Byte*>(reinterpret_cast<unsigned long long>(memory + (alignof(Params) - 1)) & -alignof(Params));

    // Params needs to be trivially destructible ... do not see how to test this in compile time (there is no equivalent to std::is_trivially_destructible in thrust as far as I can see
    Params& params = *new(memory) Params();  memory += sizeof(Params);

    params.initialize(
                      args.M,
                      args.N,
                      args.K,
                      1.,
                      args.A,
                      args.M,
                      args.B,
                      args.K,
                      .0,
                      args.C,
                      args.M,
                      args.C,
                      args.M
                      );
    
    cutlass_kernel<KernelClass><<< params.grid, params.block, sizeof(typename KernelClass::SharedStorage)>>>(params);
};


template<typename BlockShape, typename ThreadShape>
__device__ void cutlass_gemm(Mult<ut::complex> const& args, Byte*& memory)
{
    typedef cutlass::gemm::ZgemmTraits<
    cutlass::MatrixLayout::kColumnMajor,   // layout of A matrix
    cutlass::MatrixLayout::kColumnMajor,
    BlockShape,
    cutlass::gemm::LinearScaling<cuda_value_trait_t<ut::complex>>,
    ThreadShape
    >
    Traits;
    
    typedef typename Traits::Params Params;
    typedef typename Traits::KernelClass KernelClass;

    memory = reinterpret_cast<Byte*>(reinterpret_cast<unsigned long long>(memory + (alignof(Params) - 1)) & -alignof(Params));

    // Params needs to be trivially destructible ... do not see how to test this in compile time (there is no equivalent to std::is_trivially_destructible in thrust as far as I can see
    Params& params = *new(memory) Params();  memory += sizeof(Params);

    params.initialize(
                      args.M,
                      args.N,
                      args.K,
                      1.,
                      args.A,
                      args.M,
                      args.B,
                      args.K,
                      .0,
                      args.C,
                      args.M,
                      args.C,
                      args.M
                      );
    
    cutlass_kernel<KernelClass><<< params.grid, params.block, sizeof(typename KernelClass::SharedStorage)>>>(params);
};

#endif

template <typename Value>
void imp::mult(Matrix<Device, Value>& dest, Matrix<Device, Value> const& L, Matrix<Device, Value> const& R, itf::Batcher<Value>& batcher) {
    dest.I() = L.I(); dest.J() = R.J(); dest.exponent() = L.exponent() + R.exponent();
    
    auto& args = imp::get<Device>(batcher).template get_kernel<Mult<Value>>();
    
    args.A = R.data().ptr();
    args.B = L.data().ptr();
    args.C = dest.data().ptr();
    args.M = R.J();
    args.N = L.I();
    args.K = L.J();
}

//-------------------------------------------------------------------------------------------------------------------------------------------------

template <typename Value>
struct EvolveL {
    double time;
    double shift;
    double const* energies;
    cuda_value_trait_t<Value>* arg;
    int I;
    int J;
};


template <typename Value>
__global__ void kerEvolveL(EvolveL<Value> args) {
    int const i = blockIdx.x; int const j = threadIdx.x;
   
    args.arg[j + blockDim.x*i] *= exp(args.time*args.energies[i] - args.shift);
};

template <typename Value>
void imp::evolveL(Vector<Device> const& prop, Matrix<Device, Value>& arg, itf::Batcher<Value>& batcher) {
    arg.exponent() += prop.exponent();
    
    auto& args = imp::get<Device>(batcher).template get_kernel<EvolveL<Value>>();
    
    args.time     = prop.time();
    args.shift    = prop.exponent();
    args.energies = prop.energies().data().ptr();
    args.arg      = arg.data().ptr();
    args.I        = arg.I();
    args.J        = arg.J();
}

//-------------------------------------------------------------------------------------------------------------------------------------------------

#if __CUDACC_VER_MAJOR__ >= 9

template <typename Value>
__device__ __forceinline__ void reduceWarp(int const tid, cuda_value_trait_t<Value>* data, cuda_value_trait_t<Value>* result) {
    cuda_value_trait_t<Value> temp;
    temp = data[tid + 16]; __syncwarp();
    data[tid] += temp;     __syncwarp();
    temp = data[tid + 8];  __syncwarp();
    data[tid] += temp;     __syncwarp();
    temp = data[tid + 4];  __syncwarp();
    data[tid] += temp;     __syncwarp();
    temp = data[tid + 2];  __syncwarp();
    data[tid] += temp;     __syncwarp();
    temp = data[tid + 1];  __syncwarp();
    data[tid] += temp;     __syncwarp();
    if(tid == 0) *result = *data;
};

#else

template <typename Value>
__device__ __forceinline__ void reduceWarp(int const tid, cuda_value_trait_t<Value> volatile* data, cuda_value_trait_t<Value>* result) {
    data[tid] += data[tid + 16];
    data[tid] += data[tid + 8];
    data[tid] += data[tid + 4];
    data[tid] += data[tid + 2];
    data[tid] += data[tid + 1];
    if(tid == 0) *result = *data;
};

#endif

template <int Size>
__device__ __forceinline__ void reduce(int const tid, double* data, double* result) {
    if(tid < Size/2) data[tid] += data[tid + Size/2];
    __syncthreads();
    reduce<Size/2>(tid, data, result);
};

template <>
__device__ __forceinline__ void reduce<WarpSize>(int const tid, double* data, double* result) {
    if(tid < WarpSize) reduceWarp<double>(tid, data, result);
};


template <int Size>
__device__ __forceinline__ void reduce(int const tid, cuda_value_trait_t<ut::complex>* data, cuda_value_trait_t<ut::complex>* result) {
    if(tid < Size/2) data[tid] += data[tid + Size/2];
    __syncthreads();
    reduce<Size/2>(tid, data, result);
};

template <>
__device__ __forceinline__ void reduce<WarpSize>(int const tid, cuda_value_trait_t<ut::complex>* data, cuda_value_trait_t<ut::complex>* result) {
    if(tid < WarpSize) reduceWarp<ut::complex>(tid, data, result);
};

//-------------------------------------------------------------------------------------------------------------------------------------------------

template <typename Value>
struct Trace {
    cuda_value_trait_t<Value> const* arg;
    cuda_value_trait_t<Value>* result;
    int dim;
};

template <typename Value, int BlockDim>
__global__ void kerTrace(Trace<Value> args) {
    __shared__ cuda_value_trait_t<Value> cache[BlockDim + 16];   // I do not want some threads in the reduceWarp to read stuff outside the cache ... nobody of the nVidia freaks seems to care about this (and probabely they are right) but I do not see why
    cache[threadIdx.x] = .0;
    int i = threadIdx.x;
    
    while(i < args.dim) {
        cache[threadIdx.x] += args.arg[(args.dim + 1)*i];
        
        i += BlockDim;
    }
    __syncthreads();
    
    reduce<BlockDim>(threadIdx.x, cache, args.result);
};

template <typename Value>
void imp::trace(ut::Zahl<Value>* Z, ut::Zahl<Value>* accZ, Matrix<Device, Value> const& matrix, itf::Batcher<Value>& batcher) {
    auto& args = imp::get<Device>(batcher).template get_kernel<Trace<Value>>(); double exponent = matrix.exponent();
    
    args.arg    = matrix.data().ptr();
    args.result = imp::get<Device>(batcher).get_callback([=](Value buffer) { ut::Zahl<Value> temp(buffer, exponent); if(Z) *Z = temp; if(accZ) *accZ += temp;});
    args.dim    = matrix.I();
}


//-------------------------------------------------------------------------------------------------------------------------------------------------

template <typename Value>
struct TraceAtB {
    cuda_value_trait_t<Value> const* At;
    cuda_value_trait_t<Value> const* B;
    cuda_value_trait_t<Value>* result;
    int size;
};

template <typename Value, int BlockDim>
__global__ void kerTraceAtB(TraceAtB<Value> args) {
    __shared__ cuda_value_trait_t<Value> cache[BlockDim + 16];
    cache[threadIdx.x] = .0;
    int i = threadIdx.x;
    
    while(i < args.size) {
        cache[threadIdx.x] += args.At[i]*args.B[i];
        
        i += BlockDim;
    }
    __syncthreads();
    
    reduce<BlockDim>(threadIdx.x, cache, args.result);
};


template <typename Value>
void imp::traceAtB(ut::Zahl<Value>* Z, ut::Zahl<Value>* accZ, Matrix<Device, Value> const& At, Matrix<Device, Value> const& B, itf::Batcher<Value>& batcher) {
    auto& args = imp::get<Device>(batcher).template get_kernel<TraceAtB<Value>>(); double exponent = At.exponent() + B.exponent();
    
    args.At     = At.data().ptr();
    args.B      = B.data().ptr();
    args.result = imp::get<Device>(batcher).get_callback([=](Value buffer) { ut::Zahl<Value> temp(buffer, exponent); if(Z) *Z = temp; if(accZ) *accZ += temp;});
    args.size   = At.I()*At.J();
}


//-------------------------------------------------------------------------------------------------------------------------------------------------

template <typename Value>
struct Norm {
    cuda_value_trait_t<Value> const* arg;
    cuda_value_trait_t<Value>* result; // reuslt should be double -- but this breaks the callback
    int size;
};

__device__ inline double compute_norm(cuda_value_trait_t<ut::complex> value) { return (thrust::conj(value)*value).real(); }
__device__ inline double compute_norm(double value) { return value*value; }

template<typename Value, int BlockDim>
__global__ void kerNorm(Norm<Value> args) {
    __shared__ cuda_value_trait_t<Value> cache[BlockDim + 16];
    cache[threadIdx.x] = .0;
    cuda_value_trait_t<Value> value; int i = threadIdx.x;
    
    while(i < args.size) {
        value = args.arg[i];
        cache[threadIdx.x] += compute_norm(value);
        
        i += BlockDim;
    }
    __syncthreads();
    
    reduce<BlockDim>(threadIdx.x, cache, args.result);
};

template <typename Value>
void imp::norm(double* norm, Matrix<Device, Value> const& matrix, itf::Batcher<Value>& batcher) {
    auto& args = imp::get<Device>(batcher).template get_kernel<Norm<Value>>(); double exponent = matrix.exponent();
    
    args.arg    = matrix.data().ptr();
    args.result = imp::get<Device>(batcher).get_callback([=](cuda_value_trait_t<ut::complex> buffer) { *norm = std::log(thrust::abs(buffer))/2. + exponent;});
    args.size   = matrix.I()*matrix.J();
}

//-------------------------------------------------------------------------------------------------------------------------------------------------

template <typename Value>
void imp::density_matrix(Matrix<Device, Value>& dest, Matrix<Device, Value> const& B, Vector<Device> const& prop, Matrix<Device, Value> const& A, Energies<Device> const& energies, itf::Batcher<Value>& batcher)
{
    throw std::runtime_error("imp::density_matrix: not implemented !");
};

//-------------------------------------------------------------------------------------------------------------------------------------------------


template <typename Value>
struct Add {
    cuda_value_trait_t<Value> const* source;
    cuda_value_trait_t<Value>* dest;
    cuda_value_trait_t<Value>* fact; //CUDA doesn't like non-pointer thrust::complex because it has a copy constructor.
    int size;
};

template <typename Value>
__global__ void kerAdd(Add<Value> args)
{
    int const index = blockDim.x*blockIdx.x + threadIdx.x;
    
    if(index < args.size) args.dest[index] += *(args.fact)*args.source[index];
    //if(index < args.size) args.dest[index] += args.source[index];
};

template <typename Value>
void imp::add(Matrix<Device, Value>& dest, ut::Zahl<Value> const& fact, Matrix<Device, Value> const& source, itf::Batcher<Value>& batcher)
{
    auto& args = imp::get<Device>(batcher).template get_kernel<Add<Value>>();
    
    args.source    = source.data().ptr();
    args.dest      = dest.data().ptr();
    *(args.fact)   = fact.get()*ut::exp(source.exponent()).get();
    args.size      = source.I()*source.J();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Value>
using KerArgs = variant<CopyEvolveL<Value>, Mult<Value>, EvolveL<Value>, Trace<Value>, TraceAtB<Value>, Norm<Value>, Add<Value>>;

template <typename Value>
struct alignas(16) imp::Kernel {
    KerArgs<Value> args;
    int id;
};



template <typename Value>
__global__ void kerLauncher(Kernel<Value>* kernel, int const N, Byte* memory)
{
    for(int n = 0; n < N; ++n) {
        Kernel<Value>&  ker = kernel[n];
        
        if(ker.id == device::index<Mult<Value>, KerArgs<Value>>::value) {
            
            auto& args = get_device<Mult<Value>>(ker.args);
            
#ifdef HAVE_CUBLAS
            
            hipblasDgemm(deviceHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, args.M, args.N, args.K, &deviceOne, args.A, args.M, args.B, args.K, &deviceZero, args.C, args.M);

#else
            
            cutlass_gemm<cutlass::Shape<8, 64, 128>, cutlass::Shape<8, 8, 8>>(args, memory);
            
#endif
            
            
        } else if(ker.id == device::index<Norm<Value>, KerArgs<Value>>::value) {
            
            auto& args = get_device<Norm<Value>>(ker.args);
            kerNorm<Value,1024><<<1, 1024>>>(args);
            
        } else if(ker.id == device::index<EvolveL<Value>, KerArgs<Value>>::value) {
            
            auto& args = get_device<EvolveL<Value>>(ker.args);
            kerEvolveL<Value><<<args.I, args.J>>>(args);
            
        } else if(ker.id == device::index<CopyEvolveL<Value>, KerArgs<Value>>::value) {
            
            auto& args = get_device<CopyEvolveL<Value>>(ker.args);
            kerCopyEvolveL<Value><<<args.I, args.J>>>(args);
            
        } else if(ker.id == device::index<Trace<Value>, KerArgs<Value>>::value) {
            
            auto& args = get_device<Trace<Value>>(ker.args);
            kerTrace<Value,WarpSize><<<1, WarpSize>>>(args);
            
        } else if(ker.id == device::index<TraceAtB<Value>, KerArgs<Value>>::value) {
            
            auto& args = get_device<TraceAtB<Value>>(ker.args);
            kerTraceAtB<Value,1024><<<1, 1024>>>(args);
            
        } else if(ker.id == device::index<Add<Value>, KerArgs<Value>>::value) {
            
            auto& args = get_device<Add<Value>>(ker.args);
            if (args.fact == nullptr) args.fact = new cuda_value_trait_t<Value>;
            kerAdd<Value><<<(args.size + 256 - 1)/256, 256>>>(args);
            
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

imp::Batcher<Device, double>::Batcher(std::size_t size) :
size_(size),
numberOfKernels_(0),
phase_(Phase::record),
deviceKernelBuffer_(alloc->get<Kernel<double>>(size_)),
deviceCallBackBuffer_(alloc->get<double>(size_)),
memory_(alloc->get<Byte>(
#ifndef HAVE_CUBLAS
                         sizeof(typename cutlass::gemm::DgemmTraits<
                                cutlass::MatrixLayout::kColumnMajor,
                                cutlass::MatrixLayout::kColumnMajor>::Params)*size_
#else
                         8
#endif
                         ))
{
    cudaErrchk(hipStreamCreate(&stream_));
    
    cudaErrchk(hipHostMalloc(reinterpret_cast<void**>(&hostKernelBuffer_), size_*sizeof(Kernel<double>)));
    cudaErrchk(hipHostMalloc(reinterpret_cast<void**>(&hostCallBackBuffer_), size_*sizeof(double)));
}

imp::Batcher<Device, ut::complex>::Batcher(std::size_t size) :
size_(size),
numberOfKernels_(0),
phase_(Phase::record),
deviceKernelBuffer_(alloc->get<Kernel<ut::complex>>(size_)),
deviceCallBackBuffer_(alloc->get<cuda_value_trait_t<ut::complex>>(size_)),
memory_(alloc->get<Byte>(
#ifndef HAVE_CUBLAS
                         sizeof(typename cutlass::gemm::ZgemmTraits<
                                cutlass::MatrixLayout::kColumnMajor,
                                cutlass::MatrixLayout::kColumnMajor>::Params)*size_
#else
                         8
#endif
                         ))
{
    cudaErrchk(hipStreamCreate(&stream_));
    
    cudaErrchk(hipHostMalloc(reinterpret_cast<void**>(&hostKernelBuffer_), size_*sizeof(Kernel<ut::complex>)));
    cudaErrchk(hipHostMalloc(reinterpret_cast<void**>(&hostCallBackBuffer_), size_*sizeof(ut::complex)));
}

template <typename Value>
cuda_value_trait_t<Value>* imp::Batcher<Device, Value>::get_callback(std::function<void(cuda_value_trait_t<Value>)> callBack) {
    if(phase_ != Phase::record) throw std::runtime_error("imp::Batcher::get_callback");
    
    int index = callBack_.size();  callBack_.push_back(callBack);
    return deviceCallBackBuffer_.ptr() + index;
};

template <typename Value>
template <typename K>
K& imp::Batcher<Device, Value>::get_kernel() {
    if(phase_ != Phase::record) throw std::runtime_error("imp::Batcher::get_kernel");
    
    Kernel<Value>& ker = hostKernelBuffer_[numberOfKernels_++];
    ker.id = device::index<K, KerArgs<Value>>::value;
    return get_host<K>(ker.args);
};

template <typename Value>
void imp::Batcher<Device, Value>::launch() {
    if(phase_ != Phase::record) throw std::runtime_error("imp::Batcher::launch");
    
    if(numberOfKernels_) {
        cudaErrchk(hipMemcpyAsync(deviceKernelBuffer_.ptr(), hostKernelBuffer_, numberOfKernels_*sizeof(Kernel<Value>), hipMemcpyHostToDevice, stream_));
        kerLauncher<Value><<<1, 1, 0, stream_>>>(deviceKernelBuffer_.ptr(), numberOfKernels_, memory_.ptr());
        
        numberOfKernels_ = 0; phase_ = Phase::execute;
    }
};

template <typename Value>
int imp::Batcher<Device, Value>::is_ready() {
    if(phase_ == Phase::execute) {
        hipError_t quest = hipStreamQuery(stream_);
        
        if(quest == hipErrorNotReady) return 0;
        
        cudaErrchk(quest);

        if(callBack_.size()) {
            cudaErrchk(hipMemcpyAsync(hostCallBackBuffer_, deviceCallBackBuffer_.ptr(), callBack_.size()*sizeof(Value), hipMemcpyDeviceToHost, stream_));
            phase_ = Phase::finalize; return 0;
        }
    }
    
    if(phase_ == Phase::finalize) {
        hipError_t quest = hipStreamQuery(stream_);
        
        if(quest == hipErrorNotReady) return 0;
        
        cudaErrchk(quest);
        
        for(std::size_t index = 0; index < callBack_.size(); ++index) callBack_[index](hostCallBackBuffer_[index]);
        callBack_.clear();
    }
    
    phase_ = Phase::record; return 1;
};

template <typename Value>
imp::Batcher<Device, Value>::~Batcher() {
    alloc->free(memory_);
    
    alloc->free(deviceCallBackBuffer_);
    cudaErrchk(hipHostFree(hostCallBackBuffer_));
    
    alloc->free(deviceKernelBuffer_);
    cudaErrchk(hipHostFree(hostKernelBuffer_));
    
    cudaErrchk(hipStreamDestroy(stream_));
};


//explicit instantiations (double)
template struct Matrix<Device,double>;
template void imp::copyEvolveL(Matrix<Device, double>& dest, Vector<Device> const& prop, Matrix<Device, double> const& source, itf::Batcher<double>& batcher);
template void imp::mult(Matrix<Device, double>& dest, Matrix<Device, double> const& L, Matrix<Device, double> const& R, itf::Batcher<double>& batcher);
template void imp::evolveL(Vector<Device> const& prop, Matrix<Device, double>& arg, itf::Batcher<double>& batcher);
template void imp::trace(ut::Zahl<double>* Z, ut::Zahl<double>* accZ, Matrix<Device, double> const& matrix, itf::Batcher<double>& batcher);
template void imp::traceAtB(ut::Zahl<double>* Z, ut::Zahl<double>* accZ, Matrix<Device, double> const& At, Matrix<Device, double> const& B, itf::Batcher<double>& batcher);
template void imp::norm(double* norm, Matrix<Device, double> const& matrix, itf::Batcher<double>& batcher);
template void imp::add(Matrix<Device, double>& dest, ut::Zahl<double> const& fact, Matrix<Device, double> const& source, itf::Batcher<double>& batcher);
template void imp::density_matrix(Matrix<Device, double>& dest, Matrix<Device, double> const& B, Vector<Device> const& prop, Matrix<Device, double> const& A, Energies<Device> const& energies, itf::Batcher<double>& batcher);


//explicit instantiations (complex)
template struct Matrix<Device,ut::complex>;
template void imp::copyEvolveL(Matrix<Device, ut::complex>& dest, Vector<Device> const& prop, Matrix<Device, ut::complex> const& source, itf::Batcher<ut::complex>& batcher);
template void imp::mult(Matrix<Device, ut::complex>& dest, Matrix<Device, ut::complex> const& L, Matrix<Device, ut::complex> const& R, itf::Batcher<ut::complex>& batcher);
template void imp::evolveL(Vector<Device> const& prop, Matrix<Device, ut::complex>& arg, itf::Batcher<ut::complex>& batcher);
template void imp::trace(ut::Zahl<ut::complex>* Z, ut::Zahl<ut::complex>* accZ, Matrix<Device, ut::complex> const& matrix, itf::Batcher<ut::complex>& batcher);
template void imp::traceAtB(ut::Zahl<ut::complex>* Z, ut::Zahl<ut::complex>* accZ, Matrix<Device, ut::complex> const& At, Matrix<Device, ut::complex> const& B, itf::Batcher<ut::complex>& batcher);
template void imp::norm(double* norm, Matrix<Device, ut::complex> const& matrix, itf::Batcher<ut::complex>& batcher);
template void imp::add(Matrix<Device, ut::complex>& dest, ut::Zahl<ut::complex> const& fact, Matrix<Device, ut::complex> const& source, itf::Batcher<ut::complex>& batcher);
template void imp::density_matrix(Matrix<Device, ut::complex>& dest, Matrix<Device, ut::complex> const& B, Vector<Device> const& prop, Matrix<Device, ut::complex> const& A, Energies<Device> const& energies, itf::Batcher<ut::complex>& batcher);
